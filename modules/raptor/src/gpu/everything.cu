#include "motis/raptor/gpu/gpu_raptor.cuh"
#include "motis/raptor/gpu/gpu_timetable.cuh"
#include "motis/raptor/raptor_query.h"

#include <algorithm>

namespace motis::raptor {

__constant__ device_gpu_timetable GTT;

template <typename Kernel>
void inline launch_kernel(Kernel kernel, void** args,
                          device_context const& device, hipStream_t s) {
  hipSetDevice(device.id_);

  hipLaunchCooperativeKernel((void*)kernel, device.grid_,
                              device.threads_per_block_, args, 0, s);
  cc();
}

void fetch_arrivals_async(d_query const& dq, hipStream_t s) {
  hipMemcpyAsync(
      dq.mem_->host_.result_->data(), dq.mem_->device_.result_.front(),
      dq.mem_->host_.result_->byte_size(), hipMemcpyDeviceToHost, s);
  cc();
}

void fetch_arrivals_async(d_query const& dq, raptor_round const round_k,
                          hipStream_t s) {
  hipMemcpyAsync((*dq.mem_->host_.result_)[round_k],
                  dq.mem_->device_.result_[round_k],
                  dq.mem_->host_.result_->stop_count_ * sizeof(time),
                  hipMemcpyDeviceToHost, s);
  cc();
}

}  // namespace motis::raptor

#include "gpu_raptor.cu"
#include "gpu_timetable.cu"
#include "hybrid_raptor.cu"
